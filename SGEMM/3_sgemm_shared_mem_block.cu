#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define BLOCKSIZE 32

__global__ void sgemm_shared_mem_block(int M, int N, int K, float alpha, const float *A, const float *B, float beta, float *C) {
  

    const int cRow = blockIdx.x;
    const int cCol = blockIdx.y;


    __shared__ float As[BLOCKSIZE * BLOCKSIZE];
    __shared__ float Bs[BLOCKSIZE * BLOCKSIZE];
     
    const  int threadCol = threadIdx.x % BLOCKSIZE;
    const  int threadRow = threadIdx.x / BLOCKSIZE;
     
    A += cRow * BLOCKSIZE * K;                    // row=cRow, col=0
    B += cCol * BLOCKSIZE;                        // row=0, col=cCol
    C += cRow * BLOCKSIZE * N + cCol * BLOCKSIZE; // row=cRow, col=cCol

    float tmp = 0.0;
    for (int bkIdx = 0; bkIdx < K; bkIdx += BLOCKSIZE) {

        As[threadRow * BLOCKSIZE + threadCol] = A[threadRow * K + threadCol];
        Bs[threadRow * BLOCKSIZE + threadCol] = B[threadRow * N + threadCol];


        __syncthreads();
        A += BLOCKSIZE;
        B += BLOCKSIZE * N;

         
        for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx) {
            tmp += As[threadRow * BLOCKSIZE + dotIdx] * Bs[dotIdx * BLOCKSIZE + threadCol];
        }
         
        __syncthreads();
    } 
    C[threadRow * N + threadCol] = alpha * tmp + beta * C[threadRow * N + threadCol];
}

void initialize_matrix(float* mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; ++i) {
        mat[i] = static_cast<float>(rand()) / RAND_MAX;
    }
}

int main() {
    int M = 4092;
    int N = 4092;
    int K = 4092;

    float alpha = 1.0f;
    float beta = 0.0f;

    size_t size_A = M * K * sizeof(float);
    size_t size_B = K * N * sizeof(float);
    size_t size_C = M * N * sizeof(float);

    float *h_A = (float*)malloc(size_A);
    float *h_B = (float*)malloc(size_B);
    float *h_C = (float*)malloc(size_C);

    initialize_matrix(h_A, M, K);
    initialize_matrix(h_B, K, N);
    initialize_matrix(h_C, M, N);

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size_A);
    hipMalloc((void**)&d_B, size_B);
    hipMalloc((void**)&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size_C, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCKSIZE * BLOCKSIZE); // 1024 threads
    dim3 gridDim((M + BLOCKSIZE - 1) / BLOCKSIZE, (N + BLOCKSIZE - 1) / BLOCKSIZE);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    sgemm_shared_mem_block<<<gridDim, blockDim>>>(M, N, K, alpha, d_A, d_B, beta, d_C);
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);

    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    float gflops = (2.0f * M * N * K) / (elapsedTime / 1000.0f) / 1e9;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(h_A);
    free(h_B);
    free(h_C);

    printf("GFLOPs: %f\n", gflops);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
